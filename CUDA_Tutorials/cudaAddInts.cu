#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>

using namespace std;

__global__ void AddIntsCUDA(int *a, int *b)
{
	a[0] += b[0];
}

int main()
{

int h_a=5, h_b=9;
int *d_a, *d_b;

if(hipMalloc(&d_a, sizeof(int)) != hipSuccess)
{
	cout<<"Error allocating memory!"<<endl;
	return 0;
}
if(hipMalloc(&d_b, sizeof(int)) != hipSuccess)
{
	cout<<"Error allocating memory!"<<endl;
	return 0;
}

if(hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
{
	cout<<"Error copying memory!"<<endl;
	hipFree(d_a);
	hipFree(d_b);
	return 0;
}
if(hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice) != cudaSu	ccess)
{
	cout<<"Error copying memory!"<<endl;
	hipFree(d_a);
	hipFree(d_b);
	return 0;
}

AddIntsCUDA<<<1,1>>>(d_a, d_b);

if(hipMemcpy(&h_a, d_a, sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
{
	cout<<"Error copying memory!"<<endl;
	hipFree(d_a);
	hipFree(d_b);
	return 0;
}

cout<< "The answer is : "<<a<<endl;

hipFree(d_a);
hipFree(d_b);

return 0;
}
